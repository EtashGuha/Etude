#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */
 
#include <hip/hip_runtime.h>
 
#ifndef nIsPow2
#define nIsPow2 false
#endif

#ifndef blockSize
#define blockSize 256
#endif

#ifdef __DEVICE_EMULATION__
#define EMUSYNC __syncthreads()
#else
#define EMUSYNC
#endif
 
 
__device__ inline Real_t norm2(Real_t x, Real_t y) {
	return x*x + y*y;
}

__global__ void countInCircle(Real_t *g_idata, int *g_odata, unsigned int n)
{
    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.

     __shared__ volatile int sdata[2 * 256];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
     unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    int mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        mySum += norm2(g_idata[2*i], g_idata[2*i + 1]) > 1 ? 0 : 1;
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
        	 mySum += norm2(g_idata[2*(i+blockSize)], g_idata[2*(i+blockSize) + 1]) > 1 ? 0 : 1;
            
        i += gridSize;
    }

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();

   // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
    
#ifndef __DEVICE_EMULATION__
    if (tid < 32)
#endif
    {
        if (blockSize >=  64) { sdata[tid] = mySum = mySum + sdata[tid + 32]; EMUSYNC; }
        if (blockSize >=  32) { sdata[tid] = mySum = mySum + sdata[tid + 16]; EMUSYNC; }
        if (blockSize >=  16) { sdata[tid] = mySum = mySum + sdata[tid +  8]; EMUSYNC; }
        if (blockSize >=   8) { sdata[tid] = mySum = mySum + sdata[tid +  4]; EMUSYNC; }
        if (blockSize >=   4) { sdata[tid] = mySum = mySum + sdata[tid +  2]; EMUSYNC; }
        if (blockSize >=   2) { sdata[tid] = mySum = mySum + sdata[tid +  1]; EMUSYNC; }
    }

       
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];

}