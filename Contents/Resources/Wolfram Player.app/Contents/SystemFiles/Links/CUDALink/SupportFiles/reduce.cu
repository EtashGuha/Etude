#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */
 
 __global__ void reduce(T1 *g_idata, T1 *g_odata, unsigned int n)
{
    __shared__ volatile T1 sdata[2 * 256];

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;
    
    T1 mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the 
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {         
        mySum += g_idata[i];
        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n) 
            mySum += g_idata[i+blockSize];  
        i += gridSize;
    } 

    // each thread puts its local sum into shared memory 
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if (blockSize >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
    if (blockSize >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
    if (blockSize >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }
    

    // now that we are using warp-synchronous programming (below)
    // we need to declare our shared memory volatile so that the compiler
    // doesn't reorder stores to it and induce incorrect behavior.
    volatile T1* smem = sdata;
    if (blockSize >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
    if (blockSize >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
    if (blockSize >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
    if (blockSize >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
    if (blockSize >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
    if (blockSize >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    
    // write result for this block to global mem 
    if (tid == 0) 
        g_odata[blockIdx.x] = sdata[0];
}
